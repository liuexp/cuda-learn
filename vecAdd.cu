#include "hip/hip_runtime.h"
#include<cstdio>

typedef struct {
	int size;
	int v;
	int *vs;
} VertexList;

const float eps = 1e-7;

__global__ void vecAdd(float *C, float *A, float *B, int N){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<N)C[i] = A[i] + B[i];
}

int main(){
	const int n = (1<<23);
	srand(time(NULL));
	const int ns = n * sizeof(float);

	printf("%d\n",ns);
	float *A,*B,*C,*dA,*dB,*dC;
	A=(float *)malloc(ns);
	B=(float *)malloc(ns);
	C=(float *)malloc(ns);
	hipMalloc((void **)&dA, ns);
	hipMalloc((void **)&dB, ns);
	hipMalloc((void **)&dC, ns);
	for(int i=0;i<1000;i++){
		A[i]=float(rand())/RAND_MAX;
		B[i]=float(rand())/RAND_MAX;
	}
	hipMemcpy(dA, A, ns, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, ns, hipMemcpyHostToDevice);
	vecAdd<<<n/256, 256>>> (dC, dA, dB, n);
	hipMemcpy(C, dC, ns, hipMemcpyDeviceToHost);
	for(int i=0;i<100;i++){
		if(abs(C[i] - A[i] - B[i])<eps)continue;
		printf("%.3f + %.3f = %.3f\t %.3f\n", A[i], B[i], C[i], A[i] + B[i]);
	}
	gets(NULL);
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	free(A);
	free(B);
	free(C);
}
