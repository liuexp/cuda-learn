#include "hip/hip_runtime.h"
#include<cstdio>

__global__ void vecAdd(float *C, float *A, float *B){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	C[i] = A[i] + B[i];
}

int main(){
	const int n = (1<<15);
	srand(time(NULL));
	const int ns = n * sizeof(float);

	printf("%d\n",ns);
	float *A,*B,*C,*dA,*dB,*dC;
	A=(float *)malloc(ns);
	B=(float *)malloc(ns);
	C=(float *)malloc(ns);
	hipMalloc((void **)&dA, ns);
	hipMalloc((void **)&dB, ns);
	hipMalloc((void **)&dC, ns);
	for(int i=0;i<1000;i++){
		A[i]=float(rand())/RAND_MAX;
		B[i]=float(rand())/RAND_MAX;
	}
	hipMemcpy(dA, A, ns, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, ns, hipMemcpyHostToDevice);
	vecAdd<<<ns/256, 256>>> (dC, dA, dB);
	hipMemcpy(C, dC, ns, hipMemcpyDeviceToHost);
	for(int i=0;i<100;i++){
		printf("%.3f + %.3f = %.3f\n", A[i], B[i], C[i]);
	}
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	free(A);
	free(B);
	free(C);
}
