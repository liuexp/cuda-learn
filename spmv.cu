#include "hip/hip_runtime.h"
//FIXME: what if n is too large for one turn?
#include<cstdio>
#include<cstdlib>
#include<ctime>
#include<hip/hip_runtime.h>
#include<cassert>
#include "common.h"

//y<-alpha*A*x+z
template <typename IndexType, typename ValueType>
__global__ void
spmv_csr_scalar_kernel(IndexType numRows, IndexType *csrRow, IndexType *cooColIdx, ValueType *cooVal, ValueType *x, ValueType *y, ValueType *z, ValueType alpha)
{
	const IndexType thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	const IndexType grid_size = gridDim.x * blockDim.x;
	
	for(IndexType row = thread_id; row < numRows; row += grid_size)
	{
		const IndexType row_start = csrRow[row];
		const IndexType row_end   = csrRow[row+1];
		
		ValueType sum = 0;
		for (IndexType jj = row_start; jj < row_end; jj++)
		sum += cooVal[jj] * x[cooColIdx[jj]];       
		
		y[row] = alpha * sum + z[row];
	}
}

void spmv_csr_scalar(int numRows, int *csrRow, int *cooColIdx, float *cooVal, float *x, float *y, float *z, float alpha)
{
	const size_t BLOCK_SIZE = 256;
	const size_t MAX_BLOCKS = max_active_blocks(spmv_csr_scalar_kernel<int, float>, BLOCK_SIZE, (size_t) 0);
	const size_t NUM_BLOCKS = min((int)MAX_BLOCKS, (int)DIVIDE_INTO(numRows, BLOCK_SIZE));
	
	spmv_csr_scalar_kernel<int, float> <<<NUM_BLOCKS, BLOCK_SIZE>>> 
	    (numRows, csrRow, cooColIdx, cooVal, x, y, z, alpha);
}

void blockMatMult(int *cooRowHostIdx, int *cooColHostIdx, float *cooValHost, float *xHost, float *zHost, int n, int nnz, 
		int *cooRowIdx, int *cooColIdx, float *cooVal, float *x, float *y, float *z, int *csrRow,
		int xOffset, int nPerTurn){
	clock_t tt;
	hipError_t cudaStat;
	printf("starting memcpy to device\n");
	tt = clock();
	//FIXME: it's not necessary for CSR!
	//cudaStat = hipMemcpy(cooRowIdx, cooRowHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	//handleError(cudaStat);
	cudaStat = hipMemcpy(cooColIdx, cooColHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(cooVal, cooValHost, nnz * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(x, xHost, n * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(z, zHost+xOffset, nPerTurn * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	hipDeviceSynchronize();
	reportTimeRound("memcpy", tt);
	reportTime(tt0);
	printf("starting multiplication\n");
	tt = clock();

	spmv_csr_scalar(nPerTurn, csrRow, cooColIdx, cooVal, x, y, z, DAMPINGFACTOR);
	hipDeviceSynchronize();
	//cudaStat = hipMemcpy(x + xOffset, y, nPerTurn*sizeof(float), hipMemcpyDeviceToDevice);
	//handleError(cudaStat);

	reportTimeRound("multiplication",tt);
	reportTime(tt0);

	cudaStat = hipMemcpy(xHost + xOffset , y, nPerTurn * sizeof(float), hipMemcpyDeviceToHost);
	handleError(cudaStat);

}

int main(){
	int	*cooRowHostIdx, *cooColHostIdx;
	float	*cooValHost;
	float	*xHost, *zHost;
	clock_t tt;
	tt0 = clock();

	cooRowHostIdx = (int *) malloc(nnz * sizeof(int));
	cooColHostIdx = (int *) malloc(nnz * sizeof(int));
	cooValHost = (float *) malloc(nnz * sizeof(float));

	readBinMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	//readMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	for(int i=0;i<10;i++){
		printf("%d\t%d\t%.9f\n", cooRowHostIdx[i], cooColHostIdx[i], cooValHost[i]);
	}
	//readSampleMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	xHost = (float *) malloc(n * sizeof(float));
	//yHost = (float *) malloc(n * sizeof(float));
	zHost = (float *) malloc(n * sizeof(float)); // the constant vector
	for(int i=0;i<n;i++)xHost[i] = 1.0;
	for(int i=0;i<n;i++)zHost[i] = (1 - DAMPINGFACTOR) / n;
	
	int	*cooRowIdx, *cooColIdx;
	float	*cooVal;
	int	*csrRow;
	float	*x, *y, *z;
	hipError_t cudaStat;

	const unsigned int maxNNZPerTurn = min(450000000,nnz);
	const unsigned int maxNPerTurn = min(maxNNZPerTurn, n);

	//cudaStat = hipMalloc((void **)&cooRowIdx, maxNNZPerTurn * sizeof(int));
	//handleError(cudaStat);
	//FIXME: cooRowIdx is not needed for CSR.
	cooRowIdx = NULL;
	cudaStat = hipMalloc((void **)&cooColIdx, maxNNZPerTurn * sizeof(int));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&cooVal, maxNNZPerTurn * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&x, n * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&y, maxNPerTurn * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&z, maxNPerTurn * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void**)&csrRow, (maxNPerTurn+1)*sizeof(int));
	handleError(cudaStat);
	reportTime(tt0);

	int *csrRowHost = (int *) malloc(sizeof(int) * (maxNPerTurn + 1));

	// starting block operation
	// for now we group edges
	unsigned int nnzCurTurn = maxNNZPerTurn;
	for(int iter = 0;iter<niter;iter++){
		printf("---------------\n");
		printf("iteration %d starting:\n", iter);
		clock_t t_iter = clock();
		int lastRow = -1;
		for(unsigned int cooOffset = 0;cooOffset<nnz;cooOffset += maxNNZPerTurn){
			//convert to CSR
			printf("starting block operation\n");
			tt = clock();
			int j=0;
			int nCurTurn = 0;
			int xOffset = cooRowHostIdx[cooOffset];
			for(;j<maxNNZPerTurn&&j+cooOffset<nnz;nCurTurn++){
				csrRowHost[nCurTurn]=j;
				assert(nCurTurn <= maxNPerTurn);
				for(;j+cooOffset < nnz && cooRowHostIdx[cooOffset + j] <= nCurTurn + xOffset && j< maxNNZPerTurn;j++);
			}
			csrRowHost[nCurTurn] = j;
			nnzCurTurn = j;
			printf("%lld\n",(nCurTurn + 1) * sizeof(int));
			cudaStat = hipMemcpy(csrRow, csrRowHost, sizeof(int) * (nCurTurn + 1), hipMemcpyHostToDevice);
			handleError(cudaStat);
			printf("convertion to CSR done.\n");

			float partialSum = 0;
			if(lastRow == cooRowHostIdx[cooOffset])
				partialSum = xHost[lastRow];
			assert(nCurTurn + xOffset <= n+1);

			blockMatMult(cooRowHostIdx+cooOffset, cooColHostIdx+cooOffset, cooValHost+cooOffset, xHost, zHost, n, nnzCurTurn,
					cooRowIdx, cooColIdx, cooVal, x, y, z, csrRow,
					xOffset, nCurTurn);

			xHost[lastRow] += partialSum;
			lastRow = xOffset + nCurTurn - 1;
			reportTimeRound("turn(block)",tt);
			reportTime(tt0);
		}
		reportTimeRound("iteration",t_iter);
		reportTime(tt0);
	}
	dumpRes(xHost);

	free(xHost);
	free(zHost);
	free(cooColHostIdx);
	free(cooRowHostIdx);
	free(cooValHost);
	free(csrRowHost);
	
	hipFree(cooColIdx);
	hipFree(cooRowIdx);
	hipFree(cooVal);
	hipFree(csrRow);
	hipFree(x);
	hipFree(y);
	hipFree(z);


	reportTime(tt0);
	hipDeviceReset();
	return 0;
}

