#include<hipsparse.h>
#include<cstdio>
#include<cstdlib>

//const float RANDRESET = 0.15;
const float DAMPINGFACTOR = 0.85;
const char mtxFile[] = "test4";

void FIXLINE(char *s){
	int l = (int)strlen(s) - 1;
	if(s[l] == '\n')s[l] = 0;
}
void readMatrix(int *row, int *col, float *val, int m){
	FILE *fp = fopen(mtxFile,"r");
	char s[1024];
	int cnt = 0;
	while(fgets(s, 1024, fp)){
		FIXLINE(s);
		char del[] = "\t ";
		if(s[0] == '#' || s[0] == '%')
			continue;
		char *t;
		t = strtok(s, del);
		int a,b;
		float c;
		a = atoi(t);
		t = strtok(NULL, del);
		b = atoi(t);
		t = strtok(NULL,del);
		c = atof(t);
		row[cnt] = a;
		col[cnt] = b;
		val[cnt] = c;
		cnt++;
	}
	fclose(fp);
}

int main(){
	hipsparseHandle_t handle=0;
	hipsparseMatDescr_t descr=0;
	int	*cooRowHostIdx, *cooColHostIdx;
	float	*cooValHost;
	int	*cooRowIdx, *cooColIdx;
	float	*cooVal;
	int	*csrRow;
	float	*xHost, *zHost;
	float	*x, *y, *z;
	int	nnz, n;
	hipsparseStatus_t status;

	n = 23026589;
	nnz = 324874844;

	cooRowHostIdx = (int *) malloc(nnz * sizeof(int));
	cooColHostIdx = (int *) malloc(nnz * sizeof(int));
	cooValHost = (float *) malloc(nnz * sizeof(float));

	readMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	xHost = (float *) malloc(n * sizeof(float));
	//yHost = (float *) malloc(n * sizeof(float));
	zHost = (float *) malloc(n * sizeof(float)); // the constant vector
	for(int i=0;i<n;i++)xHost[i] = 1.0;
	for(int i=0;i<n;i++)zHost[i] = (1 - DAMPINGFACTOR) / n;

	hipMalloc((void **)&cooRowIdx, nnz * sizeof(int));
	hipMalloc((void **)&cooColIdx, nnz * sizeof(int));
	hipMalloc((void **)&cooVal, nnz * sizeof(float));
	hipMalloc((void **)&x, n * sizeof(float));
	hipMalloc((void **)&y, n * sizeof(float));
	hipMalloc((void **)&z, n * sizeof(float));

	hipMemcpy(cooRowIdx, cooRowHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cooColIdx, cooColHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cooColIdx, cooColHostIdx, nnz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(x, xHost, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(z, zHost, n * sizeof(float), hipMemcpyHostToDevice);


	hipsparseCreate(&handle);
	hipsparseCreateMatDescr(&descr); 
	       
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);  
	
	hipMalloc((void**)&csrRow,(n+1)*sizeof(csrRow[0]));
	hipsparseXcoo2csr(handle,cooRowIdx,nnz,n,csrRow,HIPSPARSE_INDEX_BASE_ZERO); 

	const float tmpFloat1 = 1.0;
	// for each iteration, y<-z, y<-(1-d)*M*x + y, x <- y
	hipMemcpy(y, z, nnz * sizeof(float), hipMemcpyDeviceToDevice);
	status = hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz,
				   &DAMPINGFACTOR, descr, cooVal, csrRow, cooColIdx, x, &tmpFloat1, y);

	if(HIPSPARSE_STATUS_SUCCESS != status){ //should use switch case here
		printf("meow\n");
	}
	hipMemcpy(x, y, n * sizeof(float), hipMemcpyDeviceToDevice);

	//copy the result to host
	hipMemcpy(xHost, x, n * sizeof(float), hipMemcpyDeviceToHost);


	for(int i=0;i<10;i++){
		printf("%d\t%.3f\n", i, x[i]);
	}
	return 0;
}

