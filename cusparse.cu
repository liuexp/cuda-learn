#include "hip/hip_runtime.h"
// Compile with nvcc -arch=sm_20 -O2 -lcusparse cusparse.cu
//FIXME: what if n is too large for one turn?
#include<cstdio>
#include<cstdlib>
#include<ctime>
#include<hip/hip_runtime.h>
#include<hipsparse.h>
#include "common.h"

void handleStatus(hipsparseStatus_t status){
	switch(status){
		case HIPSPARSE_STATUS_INVALID_VALUE:
			printf("invalid value");
			break;
		case HIPSPARSE_STATUS_NOT_INITIALIZED:
			printf("not initialized");
			break;
		case HIPSPARSE_STATUS_ARCH_MISMATCH:
			printf("arch mismatch");
			break;
		case HIPSPARSE_STATUS_EXECUTION_FAILED:
			printf("exe failed");
			break;
		case HIPSPARSE_STATUS_INTERNAL_ERROR:
			printf("internal error");
			break;
		case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
			printf("not supported");
			break;
		case HIPSPARSE_STATUS_ALLOC_FAILED:
			printf("alloc failed");
			break;
		case HIPSPARSE_STATUS_MAPPING_ERROR :
			printf("map error");
			break;
		case HIPSPARSE_STATUS_SUCCESS:
			printf("success\n");
			break;
		default:
				printf("unknown status\n");
			break;
	}
}

//FIXME:give n and nnz another name
//FIXME:iteration should be outside of turns?
void matMult(int *cooRowHostIdx, int *cooColHostIdx, float *cooValHost, float *xHost, float *zHost, int n, int nnz, 
		int *cooRowIdx, int *cooColIdx, float *cooVal, float *x, float *y, float *z, int *csrRow,
		int xOffset, int nPerTurn){
	clock_t tt;
	hipsparseStatus_t status;
	hipError_t cudaStat;
	hipsparseHandle_t handle=0;
	hipsparseMatDescr_t descr=0;
	printf("starting memcpy to device\n");
	tt = clock();
	cudaStat = hipMemcpy(cooRowIdx, cooRowHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(cooColIdx, cooColHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(cooVal, cooValHost, nnz * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(x, xHost, n * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(z, zHost, nPerTurn * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(y, zHost, nPerTurn * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	hipDeviceSynchronize();
	reportTimeRound("memcpy", tt);
	reportTime(tt0);

	hipsparseCreate(&handle);
	hipsparseCreateMatDescr(&descr); 
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);  
	//FIXME:conversion
	hipsparseXcoo2csr(handle,cooRowIdx,nnz,n,csrRow,HIPSPARSE_INDEX_BASE_ZERO); 

	const float tmpFloat1 = 1.0;
	printf("starting iteration\n");
	for(int i=0;i<niter;i++){
		tt = clock();
		// for each iteration, y<-z, y<-(1-d)*M*x + y, x <- y
		cudaStat = hipMemcpy(y, z, n * sizeof(float), hipMemcpyDeviceToDevice);
		handleError(cudaStat);
		status = hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz,
					   &DAMPINGFACTOR, descr, cooVal, csrRow, cooColIdx, x, &tmpFloat1, y);
		handleStatus(status);
		if(HIPSPARSE_STATUS_SUCCESS != status){ 
			printf("meow\n");
		}
		cudaStat = hipMemcpy(x, y, n * sizeof(float), hipMemcpyDeviceToDevice);
		handleError(cudaStat);
		hipDeviceSynchronize();
		reportTimeRound("iteration",tt);
		reportTime(tt0);
	}
	printf("starting copying to host\n");
	tt = clock();
	hipDeviceSynchronize();
	//FIXME:taking from y is probably faster than taking from x....
	cudaStat = hipMemcpy(xHost, y, n * sizeof(float), hipMemcpyDeviceToHost);
	handleError(cudaStat);

	reportTimeRound("memcpy", tt);
	reportTime(tt0);

	/* destroy matrix descriptor */ 
	status = hipsparseDestroyMatDescr(descr); 
	descr = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS) {
	    printf("Matrix descriptor destruction failed");
	}    
	
	/* destroy handle */
	status = hipsparseDestroy(handle);
	handle = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS) {
	    printf("CUSPARSE Library release of resources failed");
	}   
}
int main(){
	int	*cooRowHostIdx, *cooColHostIdx;
	float	*cooValHost;
	float	*xHost, *zHost;
	tt0 = clock();

	cooRowHostIdx = (int *) malloc(nnz * sizeof(int));
	cooColHostIdx = (int *) malloc(nnz * sizeof(int));
	cooValHost = (float *) malloc(nnz * sizeof(float));

	readBinMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	//readMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	//readSampleMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	xHost = (float *) malloc(n * sizeof(float));
	//yHost = (float *) malloc(n * sizeof(float));
	zHost = (float *) malloc(n * sizeof(float)); // the constant vector
	for(int i=0;i<n;i++)xHost[i] = 1.0;
	for(int i=0;i<n;i++)zHost[i] = (1 - DAMPINGFACTOR) / n;
	
	int	*cooRowIdx, *cooColIdx;
	float	*cooVal;
	int	*csrRow;
	float	*x, *y, *z;
	hipError_t cudaStat;

	int maxNPerTurn = n;
	int maxNNZPerTurn = nnz;

	cudaStat = hipMalloc((void **)&cooRowIdx, maxNNZPerTurn * sizeof(int));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&cooColIdx, maxNNZPerTurn * sizeof(int));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&cooVal, maxNNZPerTurn * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&x, n * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&y, maxNPerTurn * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&z, maxNPerTurn * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void**)&csrRow, (n+1)*sizeof(csrRow[0]));
	reportTime(tt0);


	int nCurTurn = n;
	int nnzCurTurn = nnz;

	matMult(cooRowHostIdx, cooColHostIdx, cooValHost, xHost, zHost, n, nnzCurTurn,
			cooRowIdx, cooColIdx, cooVal, x, y, z, csrRow,
			0, nCurTurn);

	dumpRes(xHost);

	free(xHost);
	free(zHost);
	free(cooColHostIdx);
	free(cooRowHostIdx);
	free(cooValHost);
	
	hipFree(cooColIdx);
	hipFree(cooRowIdx);
	hipFree(cooVal);
	hipFree(csrRow);
	hipFree(x);
	hipFree(y);
	hipFree(z);


	reportTime(tt0);
	hipDeviceReset();
	return 0;
}

