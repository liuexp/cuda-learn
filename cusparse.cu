#include "hip/hip_runtime.h"
// Compile with nvcc -arch=sm_20 -O2 -lcusparse cusparse.cu
#include<cstdio>
#include<cstdlib>
#include<ctime>
#include<hip/hip_runtime.h>
#include<hipsparse.h>

//const float RANDRESET = 0.15;
const float DAMPINGFACTOR = 0.85;
const char mtxFile[] = "/media/tmp/graphchi/data/test4";

void FIXLINE(char *s){
	int l = (int)strlen(s) - 1;
	if(s[l] == '\n')s[l] = 0;
}

void readMatrix(int *row, int *col, float *val, int m){
	FILE *fp = fopen(mtxFile,"r");
	char s[1024];
	int cnt = 0;
	clock_t tt = clock();
	while(fgets(s, 1024, fp)){
		FIXLINE(s);
		char del[] = "\t ";
		if(s[0] == '#' || s[0] == '%')
			continue;
		char *t;
		t = strtok(s, del);
		int a,b;
		float c;
		a = atoi(t);
		t = strtok(NULL, del);
		b = atoi(t);
		t = strtok(NULL,del);
		c = atof(t);
		row[cnt] = a;
		col[cnt] = b;
		val[cnt] = c;
		cnt++;
	}
	printf("Read %d lines matrix in %.3fs\n", cnt, ((double)clock() - tt)/CLOCKS_PER_SEC);
	fclose(fp);
}

int main(){
	hipsparseHandle_t handle=0;
	hipsparseMatDescr_t descr=0;
	int	*cooRowHostIdx, *cooColHostIdx;
	float	*cooValHost;
	int	*cooRowIdx, *cooColIdx;
	float	*cooVal;
	int	*csrRow;
	float	*xHost, *zHost;
	float	*x, *y, *z;
	int	nnz, n;
	hipsparseStatus_t status;

	//time_t t0, t1;
	//double diff;
	clock_t tt;

	n = 23026589;
	nnz = 324874844;

	cooRowHostIdx = (int *) malloc(nnz * sizeof(int));
	cooColHostIdx = (int *) malloc(nnz * sizeof(int));
	cooValHost = (float *) malloc(nnz * sizeof(float));

	readMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	xHost = (float *) malloc(n * sizeof(float));
	//yHost = (float *) malloc(n * sizeof(float));
	zHost = (float *) malloc(n * sizeof(float)); // the constant vector
	for(int i=0;i<n;i++)xHost[i] = 1.0;
	for(int i=0;i<n;i++)zHost[i] = (1 - DAMPINGFACTOR) / n;

	hipMalloc((void **)&cooRowIdx, nnz * sizeof(int));
	hipMalloc((void **)&cooColIdx, nnz * sizeof(int));
	hipMalloc((void **)&cooVal, nnz * sizeof(float));
	hipMalloc((void **)&x, n * sizeof(float));
	hipMalloc((void **)&y, n * sizeof(float));
	hipMalloc((void **)&z, n * sizeof(float));
	hipMalloc((void**)&csrRow,(n+1)*sizeof(csrRow[0]));

	printf("starting memcpy\n");
	tt = clock();
	hipMemcpy(cooRowIdx, cooRowHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cooColIdx, cooColHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cooVal, cooValHost, nnz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(x, xHost, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(z, zHost, n * sizeof(float), hipMemcpyHostToDevice);
	printf("memcpy done in %.3fs\n", ((double)clock() - tt)/CLOCKS_PER_SEC);


	hipsparseCreate(&handle);
	hipsparseCreateMatDescr(&descr); 
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);  
	hipsparseXcoo2csr(handle,cooRowIdx,nnz,n,csrRow,HIPSPARSE_INDEX_BASE_ZERO); 

	const float tmpFloat1 = 1.0;
	
	printf("starting iteration\n");
	tt = clock();
	// for each iteration, y<-z, y<-(1-d)*M*x + y, x <- y
	hipMemcpy(y, z, nnz * sizeof(float), hipMemcpyDeviceToDevice);
	status = hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz,
				   &DAMPINGFACTOR, descr, cooVal, csrRow, cooColIdx, x, &tmpFloat1, y);
	if(HIPSPARSE_STATUS_SUCCESS != status){ //should use switch case here
		printf("meow\n");
	}
	hipMemcpy(x, y, n * sizeof(float), hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	printf("iteration done in %.3fs\n", ((double)clock() - tt)/CLOCKS_PER_SEC);

	printf("starting copying to host\n");
	tt = clock();
	hipMemcpy(xHost, x, n * sizeof(float), hipMemcpyDeviceToHost);
	printf("memcpy done in %.3fs\n", ((double)clock() - tt)/CLOCKS_PER_SEC);

	for(int i=0;i<10;i++){
		printf("%d\t%.3f\n", i, x[i]);
	}


	free(xHost);
	free(zHost);
	free(cooColHostIdx);
	free(cooRowHostIdx);
	free(cooValHost);
	hipFree(cooColIdx);
	hipFree(cooRowIdx);
	hipFree(cooVal);
	hipFree(csrRow);
	hipFree(x);
	hipFree(y);
	hipFree(z);

	/* destroy matrix descriptor */ 
	status = hipsparseDestroyMatDescr(descr); 
	descr = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS) {
	    printf("Matrix descriptor destruction failed");
	    return 1;
	}    
	
	/* destroy handle */
	status = hipsparseDestroy(handle);
	handle = 0;
	if (status != HIPSPARSE_STATUS_SUCCESS) {
	    printf("CUSPARSE Library release of resources failed");
	    return 1;
	}   
	return 0;
}

