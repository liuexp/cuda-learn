#include "hip/hip_runtime.h"
#include<cstdio>
#include<cstdlib>
#include<ctime>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<cassert>
#include "common.h"
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> 

#define STREAM_COUNT 4
hipEvent_t cycleDone[STREAM_COUNT];
hipStream_t stream[STREAM_COUNT];
hipEvent_t start, stop;

//y<-alpha*A*x+z
template <typename IndexType, typename ValueType>
__global__ void
spmv_csr_scalar_kernel(IndexType numRows, IndexType cooOffset, IndexType *csrRow, IndexType *cooColIdx, int *outDegree, ValueType *x, ValueType *y, ValueType alpha, ValueType beta)
{
	const IndexType thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	const IndexType grid_size = gridDim.x * blockDim.x;
	//FIXME: x[col]/outDegree[col] should be done before aggregate
	for(IndexType row = thread_id; row < numRows; row += grid_size)
	{
		if(csrRow[row] < cooOffset)continue;
		const IndexType row_start = csrRow[row] - cooOffset; 	//NOTE: row_start can be unsigned so may never < 0
		const IndexType row_end   = csrRow[row+1] - cooOffset;
		
		ValueType sum = 0;
		for (IndexType jj = row_start; jj < row_end; jj++){
			IndexType col = cooColIdx[jj];
			sum += x[col] / ((float)outDegree[col]);
		}
		
		y[row] = alpha * sum + beta;
		//FIXME: +beta should be done after aggregate
	}
}

void spmv_csr_scalar(int numRows, int cooOffset, int *csrRow, int *cooColIdx, int *outDegree, float *x, float *y, float alpha, float beta)
{
	const size_t BLOCK_SIZE = 512;
	int T_BLOCKS = (int)DIVIDE_INTO(numRows, BLOCK_SIZE);
	const size_t MAX_BLOCKS = max_active_blocks(spmv_csr_scalar_kernel<int, float>, BLOCK_SIZE, (size_t) 0);
	const size_t NUM_BLOCKS = min((int)MAX_BLOCKS, T_BLOCKS);
	spmv_csr_scalar_kernel<int, float> <<<NUM_BLOCKS, BLOCK_SIZE>>> 
	    (numRows, cooOffset, csrRow, cooColIdx, outDegree, x, y, alpha, beta);
}

int main(){
	//TODO: cooColHostIdx[STREAM_COUNT];
	int	*csrHost, *cooColHostIdx;
	int	*outDegreeHost;
	float	*xHost, *yHost;

	tt0 = clock();
	time(&realt0);
	
	readMetaMatrix(&outDegreeHost, NULL, &csrHost);
	//cooColHostIdx = (int *) malloc(nnz * sizeof(int));
	handleError(hipHostAlloc(&cooColHostIdx, nnz*sizeof(int), hipHostMallocDefault));
	if(cooColHostIdx == NULL)exit(-1);


	//xHost = (float *) malloc(n * sizeof(float));
	//yHost = (float *) malloc(n * sizeof(float));
	handleError(hipHostAlloc(&xHost, nnz*sizeof(float), hipHostMallocDefault));
	handleError(hipHostAlloc(&yHost, nnz*sizeof(float), hipHostMallocDefault));
	for(unsigned int i=0;i<n;i++)yHost[i] = 1.0/n;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);


	//TODO: cooColIdx[STREAM_COUNT];
	int	*cooColIdx;
	int	*csr;
	int	*outDegree;
	float	*x, *y;

	const unsigned int maxNNZPerTurn = min(GPUMEM,nnz);
	const unsigned int maxNPerTurn = min(maxNNZPerTurn, n);

	handleError(hipMalloc((void **)&cooColIdx, maxNNZPerTurn * sizeof(int)));
	handleError(hipMalloc((void **)&outDegree, n * sizeof(int)));
	handleError(hipMalloc((void **)&x, n * sizeof(float)));
	handleError(hipMalloc((void **)&y, maxNPerTurn * sizeof(float)));
	handleError(hipMalloc((void**)&csr, (n+1)*sizeof(int)));

	handleError(hipMemcpy(csr, csrHost, sizeof(int) * (n + 1), hipMemcpyHostToDevice));
	handleError(hipMemcpy(outDegree, outDegreeHost, n * sizeof(int), hipMemcpyHostToDevice));
	reportTime(tt0);
	reportTimeReal();
	// starting block operation
	// for now we group edges
	for(int iter = 0;iter<niter;iter++){
		printf("---------------\n");
		printf("iteration %d starting:\n", iter);
		clock_t t_iter = clock();
		
		int nCurTurn, cooOffset;
		unsigned int nnzCurTurn = loadBlockMatrixCsr(cooColHostIdx, 0, nCurTurn, cooOffset);
		handleError(hipMemcpyAsync(cooColIdx, cooColHostIdx, nnzCurTurn * sizeof(int), hipMemcpyHostToDevice, 0));
		handleError(hipMemcpyAsync(x, yHost, n * sizeof(float), hipMemcpyHostToDevice, 0));
		int lastRow = -1;
		int lastPartialResult = 0;
		int curXOffset = 0;

		for(unsigned int i = 1; i < numShards ; i++){
			printf("[Turn %d] started.\n", i);
			int csrOffset = cooOffset >= csrHost[lastRow + 1] ? lastRow + 1: lastRow;
			hipEventRecord(start,0);
			//FIXME:specify a stream
			spmv_csr_scalar(nCurTurn, cooOffset, &csr[csrOffset], cooColIdx, outDegree, x, y, DAMPINGFACTOR, (1-DAMPINGFACTOR)/n);
			if(lastRow == curXOffset)
				lastPartialResult = yHost[lastRow];
			handleError(hipMemcpyAsync(&yHost[curXOffset], y, nCurTurn * sizeof(float), hipMemcpyDeviceToHost, 0));
			hipEventRecord(stop,0);
			curXOffset += nCurTurn;
			nnzCurTurn = loadBlockMatrixCsr(cooColHostIdx, i, nCurTurn, cooOffset);
			handleError(hipMemcpyAsync(cooColIdx, cooColHostIdx, nnzCurTurn * sizeof(int), hipMemcpyHostToDevice, 0));
			hipDeviceSynchronize();
			//hipEventSynchronize(stop);
			yHost[lastRow] += lastPartialResult - (1-DAMPINGFACTOR)/n;
			lastRow = curXOffset + nCurTurn -1;
			reportTimeReal();
		}

		int csrOffset = cooOffset >= csrHost[lastRow + 1] ? lastRow + 1: lastRow;
		spmv_csr_scalar(nCurTurn, cooOffset, &csr[csrOffset], cooColIdx, outDegree, x, y, DAMPINGFACTOR, (1-DAMPINGFACTOR)/n);
		if(lastRow == curXOffset)
			lastPartialResult = yHost[lastRow];
		handleError(hipMemcpyAsync(&yHost[curXOffset] , y, nCurTurn * sizeof(float), hipMemcpyDeviceToHost, 0));
		curXOffset += nCurTurn;
		hipDeviceSynchronize();
		yHost[lastRow] += lastPartialResult - (1-DAMPINGFACTOR)/n;

		//memcpy(xHost, yHost, n*sizeof(float));
		reportTimeRound("iteration",t_iter);
		reportTime(tt0);
		reportTimeReal();
	}
	//FIXME: xHost is not needed.
	memcpy(xHost, yHost, n*sizeof(float));
	dumpRes(xHost);

//	free(xHost);
//	free(yHost);
//	free(cooColHostIdx);
	hipHostFree(xHost);
	hipHostFree(yHost);
	hipHostFree(cooColHostIdx);
	free(csrHost);
	free(outDegreeHost);
	
	hipFree(cooColIdx);
	hipFree(outDegree);
	hipFree(csr);
	hipFree(x);
	hipFree(y);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	reportTime(tt0);
	reportTimeReal();
	hipDeviceReset();
	return 0;
}

