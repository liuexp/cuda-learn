#include "hip/hip_runtime.h"
//FIXME: what if n is too large for one turn?
#include<cstdio>
#include<cstdlib>
#include<ctime>
#include<hip/hip_runtime.h>
#include<cassert>
#include "common.h"

//y<-alpha*A*x+z
template <typename IndexType, typename ValueType>
__global__ void
spmv_csr_scalar_kernel(IndexType numRows, IndexType *csrRow, IndexType *cooColIdx, ValueType *cooVal, ValueType *x, ValueType *y, ValueType alpha, ValueType beta)
{
	const IndexType thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	const IndexType grid_size = gridDim.x * blockDim.x;
	
	for(IndexType row = thread_id; row < numRows; row += grid_size)
	{
		const IndexType row_start = csrRow[row];
		const IndexType row_end   = csrRow[row+1];
		
		ValueType sum = 0;
		for (IndexType jj = row_start; jj < row_end; jj++)
		sum += cooVal[jj] * x[cooColIdx[jj]];       
		
		y[row] = alpha * sum + beta;
	}
}

void spmv_csr_scalar(int numRows, int *csrRow, int *cooColIdx, float *cooVal, float *x, float *y, float alpha, float beta)
{
	const size_t BLOCK_SIZE = 256;
	const size_t MAX_BLOCKS = max_active_blocks(spmv_csr_scalar_kernel<int, float>, BLOCK_SIZE, (size_t) 0);
	int T_BLOCKS = (int)DIVIDE_INTO(numRows, BLOCK_SIZE);
	if((int)MAX_BLOCKS < T_BLOCKS)
		printf("meow!! only %d blocks available but needed %d\n", (int)MAX_BLOCKS, T_BLOCKS);
	const size_t NUM_BLOCKS = min((int)MAX_BLOCKS, T_BLOCKS);
	spmv_csr_scalar_kernel<int, float> <<<T_BLOCKS, BLOCK_SIZE>>> 
	    (numRows, csrRow, cooColIdx, cooVal, x, y, alpha, beta);
}

void blockMatMult(int *cooRowHostIdx, int *cooColHostIdx, float *cooValHost, float *xHost, float *yHost, int n, int nnz, 
		int *cooRowIdx, int *cooColIdx, float *cooVal, float *x, float *y, int *csrRow,
		int lastXOffset, int nPerTurn){
	clock_t tt;
	hipError_t cudaStat;
	printf("start loading block & memcpy to device\n");
	tt = clock();

	hipDeviceSynchronize();
	cudaStat = hipMemcpy(yHost + lastXOffset , y, nPerTurn * sizeof(float), hipMemcpyDeviceToHost);
	handleError(cudaStat);
	yHost[lastRow] += partialSum;
	lastRow = lastXOffset + nCurTurn - 1;

	cudaStat = hipMemcpy(cooColIdx, cooColHostIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(cooVal, cooValHost, nnz * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	cudaStat = hipMemcpy(x, xHost, n * sizeof(float), hipMemcpyHostToDevice);
	handleError(cudaStat);
	hipDeviceSynchronize();
	reportTimeRound("memcpy", tt);
	reportTime(tt0);

	spmv_csr_scalar(nPerTurn, csrRow, cooColIdx, cooVal, x, y, DAMPINGFACTOR, (1-DAMPINGFACTOR)/n);



}

int main(){
	int	*cooRowHostIdx, *cooColHostIdx;
	float	*cooValHost;
	float	*xHost, *yHost;
	clock_t tt;

	tt0 = clock();
	time(&realt0);
	int MAX_BLOCKS = max_active_blocks(spmv_csr_scalar_kernel<int, float>, 256, (size_t) 0);
	printf("%d\n", MAX_BLOCKS);
	cooRowHostIdx = (int *) malloc(nnz * sizeof(int));
	cooColHostIdx = (int *) malloc(nnz * sizeof(int));
	cooValHost = (float *) malloc(nnz * sizeof(float));

	//readBinMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	//readMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	readMetaMatrix();
	//for(int i=0;i<10;i++){
	//	printf("%d\t%d\t%.9f\n", cooRowHostIdx[i], cooColHostIdx[i], cooValHost[i]);
	//}

	//readSampleMatrix(cooRowHostIdx, cooColHostIdx, cooValHost, nnz);
	xHost = (float *) malloc(n * sizeof(float));
	yHost = (float *) malloc(n * sizeof(float));
	for(int i=0;i<n;i++)xHost[i] = 1.0;
	
	int	*cooRowIdx, *cooColIdx;
	float	*cooVal;
	int	*csrRow;
	float	*x, *y;
	hipError_t cudaStat;

	const unsigned int maxNNZPerTurn = min(50000000,nnz);
	const unsigned int maxNPerTurn = min(maxNNZPerTurn, n);

	//cudaStat = hipMalloc((void **)&cooRowIdx, maxNNZPerTurn * sizeof(int));
	//handleError(cudaStat);
	//FIXME: cooRowIdx is not needed for CSR.
	cooRowIdx = NULL;
	cudaStat = hipMalloc((void **)&cooColIdx, maxNNZPerTurn * sizeof(int));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&cooVal, maxNNZPerTurn * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&x, n * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void **)&y, maxNPerTurn * sizeof(float));
	handleError(cudaStat);
	cudaStat = hipMalloc((void**)&csrRow, (maxNPerTurn+1)*sizeof(int));
	handleError(cudaStat);
	reportTime(tt0);
	reportTimeReal();

	int *csrRowHost = (int *) malloc(sizeof(int) * (maxNPerTurn + 1));

	// starting block operation
	// for now we group edges
	for(int iter = 0;iter<niter;iter++){
		printf("---------------\n");
		printf("iteration %d starting:\n", iter);
		clock_t t_iter = clock();
		int lastRow = -1;
		int lastPartialResult = 0;
		memset(yHost, 0, sizeof(float)*n);
		for(unsigned int cooOffset = 0, i=0; cooOffset<nnz; i++, cooOffset += maxNNZPerTurn){
			/*
			   Pipelined procedure:
			   ====loop====
			   * load block matrix (i)
			   * convert to CSR (nnzCurTurn) -> curXOffset, nCurTurn
			   * sync
			   * merge partial results(lastRow): yHost[lastRow] += partialSum
			   * memcopy csr to device
			   * backup curLastRow's result
			   * start memcpyAsync to host(lastXOffset)
			   * start memcpyAsync to device
			   * start multiplication
			   =============

			*/
			unsigned int nnzCurTurn = loadBlockMatrix(cooColHostIdx, cooRowHostIdx, cooValHost, i);
			unsigned int nCurTurn = matCoo2Csr(cooColHostIdx, cooRowHostIdx, csrRowHost, nnzCurTurn);
			hipDeviceSynchronize();
			int curLastRow = curXOffset + nCurTurn -1;
			lastPartialResult = yHost[curLastRow];
			blockMatMult();

			//convert to CSR
	
			reportTimeReal();
		}
		memcpy(xHost, yHost, n*sizeof(float));
		reportTimeRound("iteration",t_iter);
		reportTime(tt0);
		reportTimeReal();
	}
	dumpRes(xHost);

	free(xHost);
	free(yHost);
	free(cooColHostIdx);
	free(cooRowHostIdx);
	free(cooValHost);
	free(csrRowHost);
	
	hipFree(cooColIdx);
	hipFree(cooRowIdx);
	hipFree(cooVal);
	hipFree(csrRow);
	hipFree(x);
	hipFree(y);

	reportTime(tt0);
	reportTimeReal();
	hipDeviceReset();
	return 0;
}

